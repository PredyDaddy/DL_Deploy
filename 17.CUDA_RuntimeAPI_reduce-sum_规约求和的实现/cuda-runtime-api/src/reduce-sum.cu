#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void sum_kernel(float* array, int n, float* output){
   
    int position = blockIdx.x * blockDim.x + threadIdx.x;

    // 使用 extern声明外部的动态大小共享内存，由启动核函数的第三个参数指定
    extern __shared__ float cache[]; // 这个cache 的大小为 block_size * sizeof(float)
    int block_size = blockDim.x;
    int lane       = threadIdx.x;
    float value    = 0;

    if(position < n)
        value = array[position];

    for(int i = block_size / 2; i > 0; i /= 2){ // 如何理解reduce sum 参考图片：figure/1.reduce_sum.jpg
        cache[lane] = value;
        __syncthreads();  // 等待block内的所有线程储存完毕
        if(lane < i) value += cache[lane + i];
        __syncthreads();  // 等待block内的所有线程读取完毕
    }

    if(lane == 0){
        printf("block %d value = %f\n", blockIdx.x, value);
        atomicAdd(output, value); // 由于可能动用了多个block，所以汇总结果的时候需要用atomicAdd。（注意这里的value仅仅是一个block的threads reduce sum 后的结果）
    }
}

void launch_reduce_sum(float* array, int n, float* output){

    const int nthreads = 512;
    int block_size = n < nthreads ? n : nthreads;
    int grid_size = (n + block_size - 1) / block_size;

    // 这里要求block_size必须是2的幂次
    float block_sqrt = log2(block_size);
    printf("old block_size = %d, block_sqrt = %.2f\n", block_size, block_sqrt);

    block_sqrt = ceil(block_sqrt);
    block_size = pow(2, block_sqrt);

    printf("block_size = %d, grid_size = %d\n", block_size, grid_size);
    sum_kernel<<<grid_size, block_size, block_size * sizeof(float), nullptr>>>( // 这里 
        array, n, output
    ); // 这里要开辟 block_size * sizeof(float) 这么大的共享内存，
}