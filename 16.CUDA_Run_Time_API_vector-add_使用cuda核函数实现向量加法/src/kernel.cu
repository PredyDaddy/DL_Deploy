#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(const float* a, const float* b, float* c, int ndata){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx >= ndata) return;
    /*    dims                 indexs
        gridDim.z            blockIdx.z
        gridDim.y            blockIdx.y
        gridDim.x            blockIdx.x
        blockDim.z           threadIdx.z
        blockDim.y           threadIdx.y
        blockDim.x           threadIdx.x

        Pseudo code:
        position = 0
        for i in 6:
            position *= dims[i]
            position += indexs[i]
    */
    c[idx] = a[idx] + b[idx];
}

void vector_add(const float* a, const float* b, float* c, int ndata){

    const int nthreads = 512;
    int block_size = ndata < nthreads ? ndata : nthreads;  // 如果ndata < nthreads 那block_size = ndata就够了
    int grid_size = (ndata + block_size - 1) / block_size; // 其含义是我需要多少个blocks可以处理完所有的任务
    printf("block_size = %d, grid_size = %d\n", block_size, grid_size);
    vector_add_kernel<<<grid_size, block_size, 0, nullptr>>>(a, b, c, ndata);

    // 在核函数执行结束后，通过cudaPeekAtLastError获取得到的代码，来知道是否出现错误
    // cudaPeekAtLastError和cudaGetLastError都可以获取得到错误代码
    // cudaGetLastError是获取错误代码并清除掉，也就是再一次执行cudaGetLastError获取的会是success
    // 而cudaPeekAtLastError是获取当前错误，但是再一次执行cudaPeekAtLastError或者cudaGetLastErro拿到的还是那个错
    hipError_t code = hipPeekAtLastError();
    if(code != hipSuccess){    
        const char* err_name    = hipGetErrorName(code);    
        const char* err_message = hipGetErrorString(code);  
        printf("kernel error %s:%d  test_print_kernel failed. \n  code = %s, message = %s\n", __FILE__, __LINE__, err_name, err_message);   
    }
}