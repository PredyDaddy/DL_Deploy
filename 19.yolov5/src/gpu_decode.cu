
#include <hip/hip_runtime.h>

static __device__ void affine_project(float *matrix, float x, float y, float *ox, float *oy)
{
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(
    float *predict, int num_bboxes, int num_classes, float confidence_threshold,
    float *invert_affine_matrix, float *parray, int max_objects, int NUM_BOX_ELEMENT)
{
    // 确保有足够的thread, 每一个thread处理一个bounding box
    // 如果threadId超过了bounding box的数量, 这样就不会进行后续处理, 每个预测框都敲好被处理了一次
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes){
        return;
    }

    /*
     predict是n x 85tensor输出的首地址
     pitem 就是每行的指针, pitem[0] - pitem[3] 是位置信息, pitem[4]是objness
    */
    float *pitem = predict + (num_classes + 5) * position;
    float objectness = pitem[4];
    if (objectness < confidence_threshold){
        return;
    }

    // 从这个元素开始都是confidence
    float *class_confidence = pitem + 5;
    // 这里是第一个condience, 取到数值
    float confidence = *class_confidence++;

    // for循环判断是哪个类别
    int label = 0;
    for (int i = 1; i < num_classes; i++, ++class_confidence)
    {
        if (*class_confidence > confidence)
        {   
            // 如果大了, 就更新class_confidence
            confidence = *class_confidence;
            label = i; // 取到label
        }
    }

    /*
    上面的最后算出来的condifence是class_confidence只是条件概率
    当前bounding box的 confidence(置信度) =  objectness(物体概率) x class_confidence(条件概率)
    最后拿来计算置信度的confidence是最大的class_confidence
    */
    confidence *= objectness;
    if (confidence < confidence_threshold){
        return;
    }

    /*
    这里是恢复boudingbox的操作, 需要先取出来中心点(cx, cy), width, height
    */
    float cx = *pitem++;
    float cy = *pitem++;
    float width = *pitem++;
    float height = *pitem++;
    float left = cx - width * 0.5f;
    float top = cy - height * 0.5f;
    float right = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    // affine_project(invert_affine_matrix, left,  top,    &left,  &top);
    // affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    
    /*
    atomicAdd()简介: 
    int atomicAdd(int* address, int val);
    这个函数执行的操作是将指定地址 address 处的值与 val 相加，并将结果写回 address 处。这个操作是原子性的，即不会受到并发写入的干扰，保证了数据的正确性。
    使用 atomicAdd 函数可以保证多个线程在对同一个内存地址进行写操作时，不会发生数据覆盖的问题。
    由于每个线程都会在输出中写入一个bounding box，因此需要使用原子操作确保每个线程写入的位置唯一
    */

    /*
    [count, box1, box2, box3]
    因为GPU解码是多线程的, 所以需要用count记录已经处理了多少个bounding box。
    CPU单线程不需要, GPU需要确保不会将一个检测框重复输出或者漏掉。
    atomicAdd -> count +=1 返回 old_count
    这里是对parray(output_device第一个值+1)
    */
    int index = atomicAdd(parray, 1);
    // 如果超过了1000, 这个线程就没必要处理后面的boxes
    if (index >= max_objects)  
        return;
    
    // left, top, right, bottom, confidence, class, keepflag
    float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom,
    float bleft, float btop, float bright, float bbottom)
{

    float cleft = max(aleft, bleft);
    float ctop = max(atop, btop);
    float cright = min(aright, bright);
    float cbottom = min(abottom, bbottom);

    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if (c_area == 0.0f)
        return 0.0f;

    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float *bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT)
{
/*
参数解析: 
bboxes：存储了所有待处理的检测框信息的一维数组；
max_objects：最大的输出检测框数量； 案例设置的是1000, 预计一张图不会超过1000个bounding box
threshold：用于判断两个检测框是否重叠的 IOU 阈值；
NUM_BOX_ELEMENT：每个检测框存储的元素个数
一般包含: left, top, right, bottom, confidence, class, keepflag
*/
    
    // 计算position, 超过count不用进行下面计算了
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    int count = min((int)*bboxes, max_objects);
    if (position >= count){
        return; 
    }

    /*
    重叠度高, 并且类别相同，然后是condience小于另外一个, 就删掉他
    极端情况下会有误删, 如果测试cpu map的时候, 只能采用cpu nms
    日常推理的时候, 则可以使用这个NMS
    left, top, right, bottom, confidence, class, keepflag
    */
    
    // 这里计算出来当前的指针, 在bboxes上
    float *pcurrent = bboxes + 1  + position * NUM_BOX_ELEMENT; 
    // 便利每一个bbox
    for (int i = 0; i < count; ++i){
        float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        // NMS计算需要保证类别必须相同
        if (i == position || pcurrent[5] != pitem[5] ){
            continue;
        }
        
        // 判断置信度大小, 如果比pcurrent大，干掉pcurrent
        if (pitem[4] > pcurrent[4]){
            // 如果两个一样大，保留编号小的那个
            if (pitem[4] == pcurrent[4] && i < position){
                continue;
            }
                
            // 拿前面四个信息计算IOU
            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0], pitem[1], pitem[2], pitem[3]);

            if (iou > threshold){
                pcurrent[6] = 0;  // 这里pitem跟pcurrent重合度高而且达到阈值
                return;
            }
        }
    }
}

/*
decode_kernel_invoker(
    predict_device, rows, cols - 5, confidence_threshold,
    nms_threshold, nullptr, output_device, max_objects, NUM_BOX_ELEMENT, stream);
*/
void decode_kernel_invoker(
    float *predict, int num_bboxes, int num_classes, float confidence_threshold,
    float nms_threshold, float *invert_affine_matrix, float *parray, int max_objects, int NUM_BOX_ELEMENT, hipStream_t stream)
{
    /*
    参数解析:
    predict: 预测结果, 这个就是data, 未处理未过滤的predict
    num_bboxes: 在预测结果的（n x num_classes+ 5） tensor中, 多少行就是多少个box
    num_classes: 类别数量
    confidence_threshold: 置信度阈值
    nms_threshold: nms阈值
    invert_affine_matrix: 逆矩阵的指针
    parray: 输出结果数组
    max_objects: 最大数量框, 这边设置的是1000, 只是拿来确保有足够的内存
    NUM_BOX_ELEMENT: Box的element, left, top, right, bottom, confidence, class, keepflag 一共7个
    stream： 流
    */
    // 这里是确保有足够的线程去处理每一个box, 也就是每一个预测结果，所以用num_boxxes
    // 确保每个block的线程不超过512
    auto block = num_bboxes > 512 ? 512 : num_bboxes;
    auto grid = (num_bboxes + block - 1) / block;

    /* 如果核函数有波浪线，没关系，他是正常的，你只是看不顺眼罢了 */
    decode_kernel<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, confidence_threshold,
        invert_affine_matrix, parray, max_objects, NUM_BOX_ELEMENT);

    // 这里是针对每张图的框，确保每个狂都能被线程处理
    // 同样确保每个block的线程不超过512
    block = max_objects > 512 ? 512 : max_objects;
    grid = (max_objects + block - 1) / block;
    fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}
